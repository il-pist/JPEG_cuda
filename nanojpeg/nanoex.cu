#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// EXAMPLE PROGRAM                                                           //
// just define _NJ_EXAMPLE_PROGRAM to compile this (requires NJ_USE_LIBC)    //
///////////////////////////////////////////////////////////////////////////////

// EXAMPLE
// =======
//
// A few pages below, you can find an example program that uses NanoJPEG to
// convert JPEG files into PGM or PPM. To compile it, use something like
//     gcc -O3 -D_NJ_EXAMPLE_PROGRAM -o nanojpeg nanojpeg.c
// You may also add -std=c99 -Wall -Wextra -pedantic -Werror, if you want :)
// The only thing you might need is -Wno-shift-negative-value, because this
// code relies on the target machine using two's complement arithmetic, but
// the C standard does not, even though *any* practically useful machine
// nowadays uses two's complement.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "nanojpeg.h"

int main(int argc, char* argv[]) {
    int size;
    char *buf;
    FILE *f;
	clock_t start, end;

	//printf("sizeof\n  int: %lu\n  unsigned long: %lu\n  char: %lu\n  unsigned char: %lud\n  void * : %lu\n",
	//	sizeof(int), sizeof(unsigned long), sizeof(char), sizeof(unsigned char), sizeof(void*));

	start=clock();
    if (argc < 2) {
        printf("Usage: %s <input.jpg> [<output.ppm>]\n", argv[0]);
        return 2;
    }
    f = fopen(argv[1], "rb");
    if (!f) {
        printf("Error opening the input file.\n");
        return 1;
    }
    fseek(f, 0, SEEK_END);
    size = (int) ftell(f);
    buf = (char*) malloc(size);
    fseek(f, 0, SEEK_SET);
    size = (int) fread(buf, 1, size, f);
    fclose(f);

    njInit();
    if (njDecode(buf, size)) {
        free((void*)buf);
        printf("Error decoding the input file.\n");
        return 1;
    }
    free((void*)buf);

    f = fopen((argc > 2) ? argv[2] : (njIsColor() ? "nanojpeg_out.ppm" : "nanojpeg_out.pgm"), "wb");
    if (!f) {
        printf("Error opening the output file.\n");
        return 1;
    }
    fprintf(f, "P%d\n%d %d\n255\n", njIsColor() ? 6 : 5, njGetWidth(), njGetHeight());
    fwrite(njGetImage(), 1, njGetImageSize(), f);
    fclose(f);
    njDone();

	end=clock();
	printf("\nExecution time: %f\n", double(end-start) / CLOCKS_PER_SEC);
    return 0;
}

