#include "hip/hip_runtime.h"
#include "nanojpeg.h"

///////////////////////////////////////////////////////////////////////////////
// IMPLEMENTATION SECTION                                                    //
// you may stop reading here                                                 //
///////////////////////////////////////////////////////////////////////////////

#ifdef _MSC_VER
	#define NJ_INLINE static __inline
	#define NJ_FORCE_INLINE static __forceinline
#else
	#define NJ_INLINE static inline
	#define NJ_FORCE_INLINE static inline
#endif

#if NJ_USE_LIBC
	#include <stdlib.h>
	#include <stdio.h>  // added to allow compilation
	#include <string.h>
	#define njAllocMem malloc
	#define njFreeMem  free
	#define njFillMem  memset
	#define njCopyMem  memcpy
#elif NJ_USE_WIN32
	#include <windows.h>
	#define njAllocMem(size) ((void*) LocalAlloc(LMEM_FIXED, (SIZE_T)(size)))
	#define njFreeMem(block) ((void) LocalFree((HLOCAL) block))
	NJ_INLINE void njFillMem(void* block, unsigned char value, int count) { __asm {
		mov edi, block
		mov al, value
		mov ecx, count
		rep stosb
	} }
	NJ_INLINE void njCopyMem(void* dest, const void* src, int count) { __asm {
		mov edi, dest
		mov esi, src
		mov ecx, count
		rep movsb
	} }
#else
	extern void* njAllocMem(int size);
	extern void njFreeMem(void* block);
	extern void njFillMem(void* block, unsigned char byte, int size);
	extern void njCopyMem(void* dest, const void* src, int size);
#endif

#define NSTR 4

typedef struct _nj_code {
	unsigned char bits, code;
} nj_vlc_code_t;

/// One color component (descriptor + pixel data)
typedef struct _nj_cmp {
	int cid;                ///< id del descittore della component nel SoF (1, 2, 3)
	int ssx, ssy;           ///< n blocchi per minimum compressible unit / mb (ad esempio 2x2 per 4:2:0)
	int width, height;
	int stride;             ///< = nj.mbwidth * ssx * 8 : double stride: for chroma subsampling
	int qtsel;              ///< ??? da descittore della component
	int actabsel, dctabsel;
	int dcpred;
	int *intpixels; ///< pixel data for initial file read and IDCT
	int *cuintpixels; ///< pixel data for initial file read and IDCT
	unsigned char *pixels;  ///< pixel data
	unsigned char *cupixels; ///< pixel data on device
} nj_component_t;

typedef struct _nj_ctx {
	nj_result_t error;
	int use_cuda;
	hipStream_t custreams[NSTR];
	const unsigned char *pos;
	int size;
	int length;
	int width, height;      ///< dimensione immagine in pixel
	int mbwidth, mbheight;  ///< dimensione immagine in unità di minimum coded blocks / mb (8x8, 16x16...)
	int mbsizex, mbsizey;   ///< dimensione in pixel di un Minimum Coded Block / mb: 8x8, 16x16...
	int ncomp;              ///< number of components
	nj_component_t comp[3]; ///< array of components (descriptor + pixel data)
	int qtused, qtavail;
	unsigned char qtab[4][64];
	nj_vlc_code_t vlctab[4][65536];
	int buf, bufbits;
	int block[64];          ///< TEMP un blocco temporaneo usato in njDecodeBlock
	int rstinterval;
	unsigned char *rgb;
	unsigned char *curgb;
} nj_context_t;

static nj_context_t nj; /// Unique static state struct (not multithread-safe in this state)

/// Zig-Zag pattern
static const char njZZ[64] = { 0, 1, 8, 16, 9, 2, 3, 10, 17, 24, 32, 25, 18,
11, 4, 5, 12, 19, 26, 33, 40, 48, 41, 34, 27, 20, 13, 6, 7, 14, 21, 28, 35,
42, 49, 56, 57, 50, 43, 36, 29, 22, 15, 23, 30, 37, 44, 51, 58, 59, 52, 45,
38, 31, 39, 46, 53, 60, 61, 54, 47, 55, 62, 63 };

inline bool failed(hipError_t error)
{
  if (hipSuccess == error)
    return false;

  //fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
  printf("[failed] CUDA error: %s\n", hipGetErrorString(error));
  return true;
}

NJ_FORCE_INLINE unsigned char njClip(const int x) {
	return (x < 0) ? 0 : ((x > 0xFF) ? 0xFF : (unsigned char) x);
}
__device__ __forceinline__ unsigned char njCudaClip(const int x) {
	return (x < 0) ? 0 : ((x > 0xFF) ? 0xFF : (unsigned char) x);
}


#define W1 2841
#define W2 2676
#define W3 2408
#define W5 1609
#define W6 1108
#define W7 565

__global__ void njCudaRowIDCT(int* blk, int stride, int height) {
	int x0, x1, x2, x3, x4, x5, x6, x7, x8;
	
	int x = (blockIdx.x*blockDim.x + threadIdx.x)*8; // original pixel x (8 pixel per thread)
	int y = blockIdx.y*blockDim.y + threadIdx.y; // original pixel y
	blk += stride * y + x;

	if(x < stride && y < height)
	{
		if (!((x1 = blk[4] << 11)
			| (x2 = blk[6])
			| (x3 = blk[2])
			| (x4 = blk[1])
			| (x5 = blk[7])
			| (x6 = blk[5])
			| (x7 = blk[3])))
		{
			blk[0] = blk[1] = blk[2] = blk[3] = blk[4] = blk[5] = blk[6] = blk[7] = blk[0] << 3;
			return;
		}
		x0 = (blk[0] << 11) + 128;
		x8 = W7 * (x4 + x5);
		x4 = x8 + (W1 - W7) * x4;
		x5 = x8 - (W1 + W7) * x5;
		x8 = W3 * (x6 + x7);
		x6 = x8 - (W3 - W5) * x6;
		x7 = x8 - (W3 + W5) * x7;
		x8 = x0 + x1;
		x0 -= x1;
		x1 = W6 * (x3 + x2);
		x2 = x1 - (W2 + W6) * x2;
		x3 = x1 + (W2 - W6) * x3;
		x1 = x4 + x6;
		x4 -= x6;
		x6 = x5 + x7;
		x5 -= x7;
		x7 = x8 + x3;
		x8 -= x3;
		x3 = x0 + x2;
		x0 -= x2;
		x2 = (181 * (x4 + x5) + 128) >> 8;
		x4 = (181 * (x4 - x5) + 128) >> 8;
		blk[0] = (x7 + x1) >> 8;
		blk[1] = (x3 + x2) >> 8;
		blk[2] = (x0 + x4) >> 8;
		blk[3] = (x8 + x6) >> 8;
		blk[4] = (x8 - x6) >> 8;
		blk[5] = (x0 - x4) >> 8;
		blk[6] = (x3 - x2) >> 8;
		blk[7] = (x7 - x1) >> 8;
	}
}

__global__ void njCudaColIDCT(const int* blk, unsigned char *out, int stride, int height) {
	int x0, x1, x2, x3, x4, x5, x6, x7, x8;

	int x = blockIdx.x*blockDim.x + threadIdx.x; // original pixel x
	int y = (blockIdx.y*blockDim.y + threadIdx.y)*8; // original pixel y (8 pixel per thread)
	blk += stride * y + x;
	out += stride * y + x;

	if(x < stride && y < height)
	{
		if (!((x1 = blk[stride*4] << 8)
			| (x2 = blk[stride*6])
			| (x3 = blk[stride*2])
			| (x4 = blk[stride*1])
			| (x5 = blk[stride*7])
			| (x6 = blk[stride*5])
			| (x7 = blk[stride*3])))
		{
			x1 = njCudaClip(((blk[0] + 32) >> 6) + 128);
			for (x0 = 8;  x0;  --x0) {
				*out = (unsigned char) x1;
				out += stride;
			}
			return;
		}
		x0 = (blk[0] << 8) + 8192;
		x8 = W7 * (x4 + x5) + 4;
		x4 = (x8 + (W1 - W7) * x4) >> 3;
		x5 = (x8 - (W1 + W7) * x5) >> 3;
		x8 = W3 * (x6 + x7) + 4;
		x6 = (x8 - (W3 - W5) * x6) >> 3;
		x7 = (x8 - (W3 + W5) * x7) >> 3;
		x8 = x0 + x1;
		x0 -= x1;
		x1 = W6 * (x3 + x2) + 4;
		x2 = (x1 - (W2 + W6) * x2) >> 3;
		x3 = (x1 + (W2 - W6) * x3) >> 3;
		x1 = x4 + x6;
		x4 -= x6;
		x6 = x5 + x7;
		x5 -= x7;
		x7 = x8 + x3;
		x8 -= x3;
		x3 = x0 + x2;
		x0 -= x2;
		x2 = (181 * (x4 + x5) + 128) >> 8;
		x4 = (181 * (x4 - x5) + 128) >> 8;
		*out = njCudaClip(((x7 + x1) >> 14) + 128);  out += stride;
		*out = njCudaClip(((x3 + x2) >> 14) + 128);  out += stride;
		*out = njCudaClip(((x0 + x4) >> 14) + 128);  out += stride;
		*out = njCudaClip(((x8 + x6) >> 14) + 128);  out += stride;
		*out = njCudaClip(((x8 - x6) >> 14) + 128);  out += stride;
		*out = njCudaClip(((x0 - x4) >> 14) + 128);  out += stride;
		*out = njCudaClip(((x3 - x2) >> 14) + 128);  out += stride;
		*out = njCudaClip(((x7 - x1) >> 14) + 128);
	}
}

/// ======= Originale =============
NJ_INLINE void njRowIDCT(int* blk) {
	int x0, x1, x2, x3, x4, x5, x6, x7, x8;
	if (!((x1 = blk[4] << 11)
		| (x2 = blk[6])
		| (x3 = blk[2])
		| (x4 = blk[1])
		| (x5 = blk[7])
		| (x6 = blk[5])
		| (x7 = blk[3])))
	{
		blk[0] = blk[1] = blk[2] = blk[3] = blk[4] = blk[5] = blk[6] = blk[7] = blk[0] << 3;
		return;
	}
	x0 = (blk[0] << 11) + 128;
	x8 = W7 * (x4 + x5);
	x4 = x8 + (W1 - W7) * x4;
	x5 = x8 - (W1 + W7) * x5;
	x8 = W3 * (x6 + x7);
	x6 = x8 - (W3 - W5) * x6;
	x7 = x8 - (W3 + W5) * x7;
	x8 = x0 + x1;
	x0 -= x1;
	x1 = W6 * (x3 + x2);
	x2 = x1 - (W2 + W6) * x2;
	x3 = x1 + (W2 - W6) * x3;
	x1 = x4 + x6;
	x4 -= x6;
	x6 = x5 + x7;
	x5 -= x7;
	x7 = x8 + x3;
	x8 -= x3;
	x3 = x0 + x2;
	x0 -= x2;
	x2 = (181 * (x4 + x5) + 128) >> 8;
	x4 = (181 * (x4 - x5) + 128) >> 8;
	blk[0] = (x7 + x1) >> 8;
	blk[1] = (x3 + x2) >> 8;
	blk[2] = (x0 + x4) >> 8;
	blk[3] = (x8 + x6) >> 8;
	blk[4] = (x8 - x6) >> 8;
	blk[5] = (x0 - x4) >> 8;
	blk[6] = (x3 - x2) >> 8;
	blk[7] = (x7 - x1) >> 8;
}

NJ_INLINE void njColIDCT(const int* blk, unsigned char *out, int stride) {
	int x0, x1, x2, x3, x4, x5, x6, x7, x8;
	if (!((x1 = blk[8*4] << 8)
		| (x2 = blk[8*6])
		| (x3 = blk[8*2])
		| (x4 = blk[8*1])
		| (x5 = blk[8*7])
		| (x6 = blk[8*5])
		| (x7 = blk[8*3])))
	{
		x1 = njClip(((blk[0] + 32) >> 6) + 128);
		for (x0 = 8;  x0;  --x0) {
			*out = (unsigned char) x1;
			out += stride;
		}
		return;
	}
	x0 = (blk[0] << 8) + 8192;
	x8 = W7 * (x4 + x5) + 4;
	x4 = (x8 + (W1 - W7) * x4) >> 3;
	x5 = (x8 - (W1 + W7) * x5) >> 3;
	x8 = W3 * (x6 + x7) + 4;
	x6 = (x8 - (W3 - W5) * x6) >> 3;
	x7 = (x8 - (W3 + W5) * x7) >> 3;
	x8 = x0 + x1;
	x0 -= x1;
	x1 = W6 * (x3 + x2) + 4;
	x2 = (x1 - (W2 + W6) * x2) >> 3;
	x3 = (x1 + (W2 - W6) * x3) >> 3;
	x1 = x4 + x6;
	x4 -= x6;
	x6 = x5 + x7;
	x5 -= x7;
	x7 = x8 + x3;
	x8 -= x3;
	x3 = x0 + x2;
	x0 -= x2;
	x2 = (181 * (x4 + x5) + 128) >> 8;
	x4 = (181 * (x4 - x5) + 128) >> 8;
	*out = njClip(((x7 + x1) >> 14) + 128);  out += stride;
	*out = njClip(((x3 + x2) >> 14) + 128);  out += stride;
	*out = njClip(((x0 + x4) >> 14) + 128);  out += stride;
	*out = njClip(((x8 + x6) >> 14) + 128);  out += stride;
	*out = njClip(((x8 - x6) >> 14) + 128);  out += stride;
	*out = njClip(((x0 - x4) >> 14) + 128);  out += stride;
	*out = njClip(((x3 - x2) >> 14) + 128);  out += stride;
	*out = njClip(((x7 - x1) >> 14) + 128);
}

#define njThrow(e) do { nj.error = e; return; } while (0)
#define njCheckError() do { if (nj.error) return; } while (0)

static int njShowBits(int bits) {
	unsigned char newbyte;
	if (!bits) return 0;
	while (nj.bufbits < bits) {
		if (nj.size <= 0) {
			nj.buf = (nj.buf << 8) | 0xFF;
			nj.bufbits += 8;
			continue;
		}
		newbyte = *nj.pos++;
		nj.size--;
		nj.bufbits += 8;
		nj.buf = (nj.buf << 8) | newbyte;
		if (newbyte == 0xFF) {
			if (nj.size) {
				unsigned char marker = *nj.pos++;
				nj.size--;
				switch (marker) {
					case 0x00:
					case 0xFF:
						break;
					case 0xD9: nj.size = 0; break;
					default:
						if ((marker & 0xF8) != 0xD0)
							nj.error = NJ_SYNTAX_ERROR;
						else {
							nj.buf = (nj.buf << 8) | marker;
							nj.bufbits += 8;
						}
				}
			} else
				nj.error = NJ_SYNTAX_ERROR;
		}
	}
	return (nj.buf >> (nj.bufbits - bits)) & ((1 << bits) - 1);
}

NJ_INLINE void njSkipBits(int bits) {
	if (nj.bufbits < bits)
		(void) njShowBits(bits);
	nj.bufbits -= bits;
}

NJ_INLINE int njGetBits(int bits) {
	int res = njShowBits(bits);
	njSkipBits(bits);
	return res;
}

NJ_INLINE void njByteAlign(void) {
	nj.bufbits &= 0xF8;
}

static void njSkip(int count) {
	nj.pos += count;
	nj.size -= count;
	nj.length -= count;
	if (nj.size < 0) nj.error = NJ_SYNTAX_ERROR;
}

NJ_INLINE unsigned short njDecode16(const unsigned char *pos) {
	return (pos[0] << 8) | pos[1];
}

static void njDecodeLength(void) {
	if (nj.size < 2) njThrow(NJ_SYNTAX_ERROR);
	nj.length = njDecode16(nj.pos);
	if (nj.length > nj.size) njThrow(NJ_SYNTAX_ERROR);
	njSkip(2);
}

NJ_INLINE void njSkipMarker(void) {
	njDecodeLength();
	njSkip(nj.length);
}

NJ_INLINE void njDecodeSOF(void) {
	int i, ssxmax = 0, ssymax = 0; ///< ssxmax, ssymax: massimo numero di blocchi per minimum coded unit / mb (dimensione della grid più larga che si trova nell'immagine, in unità di 8x8)
	nj_component_t* c;
	njDecodeLength();
	njCheckError();
	if (nj.length < 9) njThrow(NJ_SYNTAX_ERROR);
	if (nj.pos[0] != 8) njThrow(NJ_UNSUPPORTED);
	nj.height = njDecode16(nj.pos+1);
	nj.width = njDecode16(nj.pos+3);
	if (!nj.width || !nj.height) njThrow(NJ_SYNTAX_ERROR);
	nj.ncomp = nj.pos[5];
	njSkip(6);
	switch (nj.ncomp) {
		case 1:
		case 3:
			break;
		default:
			njThrow(NJ_UNSUPPORTED);
	}
	if (nj.length < (nj.ncomp * 3)) njThrow(NJ_SYNTAX_ERROR);
	for (i = 0, c = nj.comp;  i < nj.ncomp;  ++i, ++c) { // per ogni descrittore di componente (Y/Cb/Cr/R/G/B)
		c->cid = nj.pos[0]; // pos[1] è un id incrementale del descrittore di componente
		if (!(c->ssx = nj.pos[1] >> 4)) njThrow(NJ_SYNTAX_ERROR); // pos[1] nibble superiore = n blocchi in orizzontale per minimum compressible unit / mb (ad esempio 2x2 per 4:2:0)
		if (c->ssx & (c->ssx - 1)) njThrow(NJ_UNSUPPORTED);  // non-power of two
		if (!(c->ssy = nj.pos[1] & 15)) njThrow(NJ_SYNTAX_ERROR); // pos[1] nibble inferiore = n blocchi in verticale per minimum compressible unit / mb (ad esempio 2x2 per 4:2:0)
		if (c->ssy & (c->ssy - 1)) njThrow(NJ_UNSUPPORTED);  // non-power of two
		if ((c->qtsel = nj.pos[2]) & 0xFC) njThrow(NJ_SYNTAX_ERROR);
		njSkip(3);
		nj.qtused |= 1 << c->qtsel;
		if (c->ssx > ssxmax) ssxmax = c->ssx;
		if (c->ssy > ssymax) ssymax = c->ssy;
	}
	if (nj.ncomp == 1) {
		c = nj.comp;
		c->ssx = c->ssy = ssxmax = ssymax = 1;
	}
	nj.mbsizex = ssxmax << 3;
	nj.mbsizey = ssymax << 3;
	nj.mbwidth = (nj.width + nj.mbsizex - 1) / nj.mbsizex;
	nj.mbheight = (nj.height + nj.mbsizey - 1) / nj.mbsizey;
	for (i = 0, c = nj.comp;  i < nj.ncomp;  ++i, ++c) {
		c->width = (nj.width * c->ssx + ssxmax - 1) / ssxmax;
		c->height = (nj.height * c->ssy + ssymax - 1) / ssymax;
		c->stride = nj.mbwidth * c->ssx << 3;
		if (((c->width < 3) && (c->ssx != ssxmax)) || ((c->height < 3) && (c->ssy != ssymax))) njThrow(NJ_UNSUPPORTED);
		if (!(c->pixels = (unsigned char*) njAllocMem(c->stride * nj.mbheight * c->ssy << 3))) njThrow(NJ_OUT_OF_MEM);
		if (!(c->intpixels = (int*) njAllocMem(sizeof(int) * c->stride * nj.mbheight * c->ssy << 3))) njThrow(NJ_OUT_OF_MEM);
	}
	if (nj.ncomp == 3) {
		nj.rgb = (unsigned char*) njAllocMem(nj.width * nj.height * nj.ncomp);
		if (!nj.rgb) njThrow(NJ_OUT_OF_MEM);
	}
	njSkip(nj.length);
}

/// Decode "Define Huffman Table" marker
NJ_INLINE void njDecodeDHT(void) {
	int codelen, currcnt, remain, spread, i, j;
	nj_vlc_code_t *vlc;
	static unsigned char counts[16];
	njDecodeLength();
	njCheckError();
	while (nj.length >= 17) {
		i = nj.pos[0];
		if (i & 0xEC) njThrow(NJ_SYNTAX_ERROR);
		if (i & 0x02) njThrow(NJ_UNSUPPORTED);
		i = (i | (i >> 3)) & 3;  // combined DC/AC + tableid value
		for (codelen = 1;  codelen <= 16;  ++codelen)
			counts[codelen - 1] = nj.pos[codelen];
		njSkip(17);
		vlc = &nj.vlctab[i][0];
		remain = spread = 65536;
		for (codelen = 1;  codelen <= 16;  ++codelen) {
			spread >>= 1;
			currcnt = counts[codelen - 1];
			if (!currcnt) continue;
			if (nj.length < currcnt) njThrow(NJ_SYNTAX_ERROR);
			remain -= currcnt << (16 - codelen);
			if (remain < 0) njThrow(NJ_SYNTAX_ERROR);
			for (i = 0;  i < currcnt;  ++i) {
				unsigned char code = nj.pos[i];
				for (j = spread;  j;  --j) {
					vlc->bits = (unsigned char) codelen;
					vlc->code = code;
					++vlc;
				}
			}
			njSkip(currcnt);
		}
		while (remain--) {
			vlc->bits = 0;
			++vlc;
		}
	}
	if (nj.length) njThrow(NJ_SYNTAX_ERROR);
}

/// Decode "Define Quantization Table" marker
NJ_INLINE void njDecodeDQT(void) {
	int i;
	unsigned char *t;
	njDecodeLength();
	njCheckError();
	while (nj.length >= 65) {
		i = nj.pos[0];
		if (i & 0xFC) njThrow(NJ_SYNTAX_ERROR);
		nj.qtavail |= 1 << i;
		t = &nj.qtab[i][0];
		for (i = 0;  i < 64;  ++i)
			t[i] = nj.pos[i + 1];
		njSkip(65);
	}
	if (nj.length) njThrow(NJ_SYNTAX_ERROR);
}

/// Decode "Define Restart Interval" marker
NJ_INLINE void njDecodeDRI(void) {
	njDecodeLength();
	njCheckError();
	if (nj.length < 2) njThrow(NJ_SYNTAX_ERROR);
	nj.rstinterval = njDecode16(nj.pos);
	njSkip(nj.length);
}

// Get Variable Length Code (VLC): decodes Huffman compression
static int njGetVLC(nj_vlc_code_t* vlc, unsigned char* code) {
	int value = njShowBits(16);
	int bits = vlc[value].bits;
	if (!bits) { nj.error = NJ_SYNTAX_ERROR; return 0; }
	njSkipBits(bits); // the correct number of bits for the code are consumed, even though 
	value = vlc[value].code;
	if (code) *code = (unsigned char) value;
	bits = value & 15;
	if (!bits) return 0;
	value = njGetBits(bits);
	if (value < (1 << (bits - 1)))
		value += ((-1) << bits) + 1;
	return value;
}

/// Read a block: Huffman decoding and zigzag only, to be followed by CUDA Row/ColIDCT
NJ_INLINE void njReadBlock(nj_component_t* c, int* out) {
	unsigned char code = 0, bx = 0, by = 0;
	int value, coef = 0;
	njFillMem(nj.block, 0, sizeof(nj.block)); // zero 8x8 block (OSS. only values !=0 are written)
	c->dcpred += njGetVLC(&nj.vlctab[c->dctabsel][0], NULL);
	//printf("njGetVLC (init) DC: c->dcpred=%02x\n", c->dcpred);
	nj.block[0] = (c->dcpred) * nj.qtab[c->qtsel][0];
	do {
		value = njGetVLC(&nj.vlctab[c->actabsel][0], &code);
		//printf("njGetVLC: value=%3d code=%3d; ", value, code);
		if (!code) break;  // EOB
		if (!(code & 0x0F) && (code != 0xF0)) njThrow(NJ_SYNTAX_ERROR);
		coef += (code >> 4) + 1;
		//printf("coef=%2d; ", coef);
		if (coef > 63) njThrow(NJ_SYNTAX_ERROR);

		//printf("i_block(zz)=%2d, val dequant=%d\n", njZZ[coef], value * nj.qtab[c->qtsel][coef]);
		nj.block[(int) njZZ[coef]] = value * nj.qtab[c->qtsel][coef]; // to copy directly to the output vector, njZZ (in [0:63]) would need to be njZZ_x and njZZ_y (both in [0:7])
	} while (coef < 63);
	for(coef=0, by=0; by<8; by++) // copy to output vector
	{
		for(bx=0; bx<8; bx++)
		{
			//printf("out copy bx=%d, by=%d: out=%08lx, out[%d] = nj.block[%d]\n", bx, by, (unsigned long) out, (by * c->stride + bx), coef);
			out[by * c->stride + bx] = nj.block[coef]; // [by * 8 + bx];
			coef++;
		}
	}
	// for (coef = 0;  coef < 64;  coef += 8)
	// 	njRowIDCT(&nj.block[coef]);
	// for (coef = 0;  coef < 8;  ++coef)
	// 	njColIDCT(&nj.block[coef], &out[coef], c->stride);
}

/// Decode a block: Huffman decoding, zigzag, de-quantization, iDCT (row, col)
NJ_INLINE void njDecodeBlock(nj_component_t* c, unsigned char* out) {
	unsigned char code = 0;
	int value, coef = 0;
	njFillMem(nj.block, 0, sizeof(nj.block));
	c->dcpred += njGetVLC(&nj.vlctab[c->dctabsel][0], NULL);
	nj.block[0] = (c->dcpred) * nj.qtab[c->qtsel][0];
	do {
		value = njGetVLC(&nj.vlctab[c->actabsel][0], &code);
		if (!code) break;  // EOB
		if (!(code & 0x0F) && (code != 0xF0)) njThrow(NJ_SYNTAX_ERROR);
		coef += (code >> 4) + 1;
		if (coef > 63) njThrow(NJ_SYNTAX_ERROR);
		nj.block[(int) njZZ[coef]] = value * nj.qtab[c->qtsel][coef];
	} while (coef < 63);
	for (coef = 0;  coef < 64;  coef += 8)
		njRowIDCT(&nj.block[coef]);
	for (coef = 0;  coef < 8;  ++coef)
		njColIDCT(&nj.block[coef], &out[coef], c->stride);
}

/// Read and decompress whole image (all blocks)
/// TODO separare in 4 stream, lanciare in parallelo tutti durante lettura (x3 componenti?)
NJ_INLINE void njCudaDecodeScan(void) {
	int i, mbx, mby, sbx, sby, stream_mby, stream_i, stream_n_mcb;
	dim3 dimBlock, dimGrid;
	int rstcount = nj.rstinterval, nextrst = 0;
	nj_component_t* c;
	njDecodeLength();
	njCheckError();
	if (nj.length < (4 + 2 * nj.ncomp)) njThrow(NJ_SYNTAX_ERROR);
	if (nj.pos[0] != nj.ncomp) njThrow(NJ_UNSUPPORTED);
	njSkip(1);
	for (i = 0, c = nj.comp;  i < nj.ncomp;  ++i, ++c) {
		if (nj.pos[0] != c->cid) njThrow(NJ_SYNTAX_ERROR);
		if (nj.pos[1] & 0xEE) njThrow(NJ_SYNTAX_ERROR);
		c->dctabsel = nj.pos[1] >> 4;
		c->actabsel = (nj.pos[1] & 1) | 2;
		njSkip(2);
	}
	if (nj.pos[0] || (nj.pos[1] != 63) || nj.pos[2]) njThrow(NJ_UNSUPPORTED);
	njSkip(nj.length);

	printf("Starting njCudaDecodeScan..........................\n");
	if(failed(hipDeviceSynchronize())) // ==================================
		printf("sync after ColIDCT component %d failed.\n", i);
	for(i=0; i<nj.ncomp; i++)
	{
		c = &(nj.comp[i]);
		
		printf("(DISABLED) doing hipHostRegister intpixels component %d ...\n", i);
		printf("size of locked memory: %d bytes\n", (int) (sizeof(int) * c->stride * nj.mbheight * c->ssy << 3));
		//if(failed(hipHostRegister(c->intpixels, sizeof(int) * c->stride * nj.mbheight * c->ssy << 3, hipHostRegisterDefault)))
		//	printf("hipHostRegister intpixels component %d failed\n", i);

		printf("doing malloc cuintpixels component %d ...\n", i);
		if(failed(hipMalloc((void**)&(c->cuintpixels), sizeof(int) * c->stride * nj.mbheight * c->ssy << 3))) // copy to GPU for IDFT
			printf("malloc cuintpixels component %d failed\n", i);
		
		printf("doing malloc cupixels component %d ...\n", i);
		if(failed(hipMalloc((void**)&(c->cupixels), c->stride * nj.mbheight * c->ssy << 3))) // alloc cupixels for IDFT results
			printf("malloc cupixels component %d failed\n", i);
		//if(failed(hipMemcpy( c->cupixels, c->pixels, c->stride * nj.mbheight * c->ssy << 3, hipMemcpyHostToDevice )))
		//	printf("memcpy iniziale component failed\n");

		//if (!(c->intpixels = (int*) njAllocMem(sizeof(int) * c->stride * nj.mbheight * c->ssy << 3))) njThrow(NJ_OUT_OF_MEM); // allocata altrove prima
	}
	stream_n_mcb = (nj.mbheight + NSTR-1)/NSTR; // vertical MCBs per stream
	stream_i = 0;
	stream_mby = 0;
	for (mbx = mby = 0;;) { // for each block (minimum coded unit, o minimum block: 8x8 o 16x16 o altri)
		for (i = 0, c = nj.comp;  i < nj.ncomp;  ++i, ++c) // for each component in the image (Y,Cb,Cr)
			for (sby = 0;  sby < c->ssy;  ++sby)           // for each block in the minimum coded unit
				for (sbx = 0;  sbx < c->ssx;  ++sbx) {     // es. 1x1 normalmente, o 2x2 per Cb e Cr in 4:2:0
					//printf("readblock mbx=%4d mby=%4d, component %d sbx=%d sby=%d\n", mbx, mby, i, sbx, sby);
					njReadBlock(c, &(c->intpixels[((mby * c->ssy + sby) * c->stride + mbx * c->ssx + sbx) << 3]));
					njCheckError();
				}
		if (++mbx >= nj.mbwidth) {
			mbx = 0;
			mby++;
			stream_mby++;

			if(stream_mby >= stream_n_mcb || mby >= nj.mbheight)
			{
				// start row/col IDCT (on all components) ---- FOR THIS STREAM: 1/NSTR of the whole height ----
				for(i=0; i<nj.ncomp; i++)
				{
					c = &(nj.comp[i]);
					printf("  ==== starting IDCT part %d (made of %d vertical MCBs) component %d ====\n", stream_i, stream_mby, i);

					// TODO async
					printf("component %d: memcpy cuintpix          %08lx intpix          %08lx\n", i, (unsigned long) c->cuintpixels, (unsigned long) c->intpixels);
					printf("component %d: memcpy cuintpix w/offset %08lx intpix w/offset %08lx\n", i,
						(unsigned long) ((c->cuintpixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3)),
						(unsigned long) ((c->intpixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3)));
					if(failed(hipMemcpy(      // OSS. advance memory pointers to only pick MCBs belonging to this stream
						(c->cuintpixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3), // stream_i * stream_n_mcb == height raggiunta
						(c->intpixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3),   // without sizeof(int), already intptr
						sizeof(int) * c->stride * stream_mby * c->ssy << 3, // only copy MCBs of this stream
						hipMemcpyHostToDevice )))
						printf("memcpy cuintpixels component %d failed\n", i);
					
					//if(failed(hipDeviceSynchronize())) // ================================== we want async copy
					//	printf("sync after memcpy cuintpixels component %d failed.\n", i);

					
					//if(failed(hipDeviceSynchronize())) // ==================================
					//	printf("sync after UpsampleH component %d failed.\n", i);
					printf("component %d: row cuintpix          %08lx\n", i, (unsigned long) c->cuintpixels);
					printf("component %d: row cuintpix w/offset %08lx\n", i, (unsigned long) ((c->cuintpixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3)));

					dimBlock = dim3 (4, 32);	// thread per grid cell (block): 4x32=128 thread per block (32x32 pixel elaborati)
					dimGrid = dim3 (((c->stride+7)/8 + 3)/4, ((stream_mby * c->ssy << 3) /*c->height*/+31)/32); // grid size (accounting for CUDA block size, and the 8 pixel per thread treated by RowIDCT)
					njCudaRowIDCT<<<dimGrid, dimBlock>>>(
						(c->cuintpixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3),
						c->stride,
						(stream_mby * c->ssy << 3) /*c->height*/); // height: only MCBs of this stream

					if (failed(hipPeekAtLastError()))
						printf("error RowIDCT component %d failed\n", i);
					
					printf("component %d: col cuintpix          %08lx cupix          %08lx\n", i, (unsigned long) c->cuintpixels, (unsigned long) c->cupixels);
					printf("component %d: col cuintpix w/offset %08lx cupix w/offset %08lx\n", i,
						(unsigned long) ((c->cuintpixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3)),
						(unsigned long) ((c->cupixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3)));
					dimBlock = dim3 (32, 4);	// thread per grid cell (block): 32x4=128 thread per block (32x32 pixel elaborati)
					dimGrid = dim3 ((c->stride + 31)/32, (((stream_mby * c->ssy << 3) /*c->height*/+7)/8 + 3)/4); // grid size (accounting for CUDA block size, and the 8 vertical pixel per thread treated by ColIDCT)
					njCudaColIDCT<<<dimGrid, dimBlock>>>(
						(c->cuintpixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3),
						(c->cupixels) + (stream_i * stream_n_mcb * c->stride * c->ssy << 3),
						c->stride,
						(stream_mby * c->ssy << 3) /*c->height*/); // stream_n_mcb (or less for last stream)

					if (failed(hipPeekAtLastError()))
						printf("error ColIDCT component %d failed\n", i);
					if(failed(hipDeviceSynchronize())) // ==================================
						printf("sync after ColIDCT component %d failed.\n", i);
					if(failed(hipFree(c->cuintpixels)))
						printf("free cuintpixels after IDCT component %d failed\n", i);
				}
				

				stream_mby=0;
				stream_i++;
			}
			if (mby >= nj.mbheight) break;
		}
		if (nj.rstinterval && !(--rstcount)) {
			njByteAlign();
			i = njGetBits(16);
			if (((i & 0xFFF8) != 0xFFD0) || ((i & 7) != nextrst)) njThrow(NJ_SYNTAX_ERROR);
			nextrst = (nextrst + 1) & 7;
			rstcount = nj.rstinterval;
			for (i = 0;  i < 3;  ++i)
				nj.comp[i].dcpred = 0;
		}
	}

	// TODO probabilmente CudaDeviceSynchronize?

	if(failed(hipDeviceSynchronize())) // ================================== we want async copy
		printf("sync after memcpy cuintpixels component %d failed.\n", i);

	for(i=0; i<nj.ncomp; i++)
	{
		c = &(nj.comp[i]);

		if(failed(hipHostUnregister(c->intpixels)))
			printf("hipHostUnregister intpixels component %d failed\n", i);
	}
	
	/*
	for (mbx = mby = 0;;) { // for each block (minimum coded unit, o minimum block: 8x8 o 16x16 o altri)
		for (i = 0, c = nj.comp;  i < nj.ncomp;  ++i, ++c) // for each component in the image (Y,Cb,Cr)
			for (sby = 0;  sby < c->ssy;  ++sby)           // for each block in the minimum coded unit
				for (sbx = 0;  sbx < c->ssx;  ++sbx) {     // es. 1x1 normalmente, o 2x2 per Cb e Cr in 4:2:0
					njDecodeBlock(c, &c->pixels[((mby * c->ssy + sby) * c->stride + mbx * c->ssx + sbx) << 3]);
					njCheckError();
				}
		if (++mbx >= nj.mbwidth) {
			mbx = 0;
			if (++mby >= nj.mbheight) break;
		}
		if (nj.rstinterval && !(--rstcount)) {
			njByteAlign();
			i = njGetBits(16);
			if (((i & 0xFFF8) != 0xFFD0) || ((i & 7) != nextrst)) njThrow(NJ_SYNTAX_ERROR);
			nextrst = (nextrst + 1) & 7;
			rstcount = nj.rstinterval;
			for (i = 0;  i < 3;  ++i)
				nj.comp[i].dcpred = 0;
		}
	}
	*/
	nj.error = __NJ_FINISHED;
}

/// Read and decompress whole image (all blocks)
NJ_INLINE void njDecodeScan(void) {
	int i, mbx, mby, sbx, sby;
	int rstcount = nj.rstinterval, nextrst = 0;
	nj_component_t* c;
	njDecodeLength();
	njCheckError();
	if (nj.length < (4 + 2 * nj.ncomp)) njThrow(NJ_SYNTAX_ERROR);
	if (nj.pos[0] != nj.ncomp) njThrow(NJ_UNSUPPORTED);
	njSkip(1);
	for (i = 0, c = nj.comp;  i < nj.ncomp;  ++i, ++c) {
		if (nj.pos[0] != c->cid) njThrow(NJ_SYNTAX_ERROR);
		if (nj.pos[1] & 0xEE) njThrow(NJ_SYNTAX_ERROR);
		c->dctabsel = nj.pos[1] >> 4;
		c->actabsel = (nj.pos[1] & 1) | 2;
		njSkip(2);
	}
	if (nj.pos[0] || (nj.pos[1] != 63) || nj.pos[2]) njThrow(NJ_UNSUPPORTED);
	njSkip(nj.length);
	for (mbx = mby = 0;;) { // for each block (minimum coded unit, o minimum block: 8x8 o 16x16 o altri)
		for (i = 0, c = nj.comp;  i < nj.ncomp;  ++i, ++c) // for each component in the image (Y,Cb,Cr)
			for (sby = 0;  sby < c->ssy;  ++sby)           // for each block in the minimum coded unit
				for (sbx = 0;  sbx < c->ssx;  ++sbx) {     // es. 1x1 normalmente, o 2x2 per Cb e Cr in 4:2:0
					njDecodeBlock(c, &c->pixels[((mby * c->ssy + sby) * c->stride + mbx * c->ssx + sbx) << 3]);
					njCheckError();
				}
		if (++mbx >= nj.mbwidth) {
			mbx = 0;
			if (++mby >= nj.mbheight) break;
		}
		if (nj.rstinterval && !(--rstcount)) {
			njByteAlign();
			i = njGetBits(16);
			if (((i & 0xFFF8) != 0xFFD0) || ((i & 7) != nextrst)) njThrow(NJ_SYNTAX_ERROR);
			nextrst = (nextrst + 1) & 7;
			rstcount = nj.rstinterval;
			for (i = 0;  i < 3;  ++i)
				nj.comp[i].dcpred = 0;
		}
	}
	nj.error = __NJ_FINISHED;
}

#if NJ_CHROMA_FILTER

#define CF4A (-9)
#define CF4B (111)
#define CF4C (29)
#define CF4D (-3)
#define CF3A (28)
#define CF3B (109)
#define CF3C (-9)
#define CF3X (104)
#define CF3Y (27)
#define CF3Z (-3)
#define CF2A (139)
#define CF2B (-11)
#define CUCF(x) njCudaClip(((x) + 64) >> 7) // CUDA version, later this is redefined with the non-CUDA version
#define CF(x) njClip(((x) + 64) >> 7) // non-CUDA version

/// Made to be called one thread every 4 horizontal input pixels;
///   each thread produces 8 horizontal pixels.
/// It is advantageous to avoid divergence: make 2D blocks with x=1 so first & last column
///   fall all in one block. Or at worst 2x16?
/// @param[in]  width  width of input component, not multiple of 8 / possibly halved
/// @param[in]  height height of input component, not multiple of 8 / possibly halved
/// @param[in]  stride real width of input component pixels, multiple of 8 (if applying for the first time to component, else =width)
/// @param[in]  lin    component->cupixels (original size: at least  stride*height)
/// @param[out] lout   component->cupixels (double the width:        width*height*2)
__global__ void njCudaUpsampleH(unsigned char* lin, unsigned char* lout, int width, int height, int stride) {
	//const int xmax = c->width - 3;
	int x = (blockIdx.x*blockDim.x + threadIdx.x)*4; // original pixel x
	int y = blockIdx.y*blockDim.y + threadIdx.y; // original pixel y
	int iin = stride*y+x;
	int iout = (stride*y+x) << 1; // TODO questa è width no? Altrove?
	int i;
	//printf("UpsampleH x=%d y=%d, w=%d, h=%d, str=%d, in %08lx out %08lx\n", x, y, width, height, stride, (unsigned long) lin, (unsigned long) lout);
	if(y < height)
	{
		for(i=0; i<4 && x+i<width; i++, iin+=1, iout+=2) // elaborate (4px in, 8px out) for each thread, stopping at the end of img
		{
			//if(iout+1 >= width*height*2) // TODO rimuovere
			//	printf("UpsampleH iout %d out of bounds x=%d y=%d, w=%d, h=%d, str=%d, in %08lx out %08lx\n", iout, x, y, width, height, stride, (unsigned long) lin, (unsigned long) lout);
			//if(iin >= stride*height)
			//	printf("UpsampleH iin %d out of bounds x=%d y=%d, w=%d, h=%d, str=%d, in %08lx out %08lx\n", iin, x, y, width, height, stride, (unsigned long) lin, (unsigned long) lout);

			if(x+i == 0) // first pixel (*000)
			{
				lout[iout+0] = CUCF(CF2A * lin[iin+0] + CF2B * lin[iin+1]);                         // (offset = -2 ?)
				lout[iout+1] = CUCF(CF3X * lin[iin+0] + CF3Y * lin[iin+1] + CF3Z * lin[iin+2]);     // (offset = -1 ?)
			}
			else if(x+i == 1) // second pixel (0*00)
			{
				lout[iout+0] = CUCF(CF3A * lin[iin-1] + CF3B * lin[iin+0] + CF3C * lin[iin+1]);     // (offset = -1 ?)
				if(x+i == width-2) // second pixel is also second to last (0*0) (image ends right after it started: 3 column wide)
					lout[iout+1] = CUCF(CF3A * lin[iin+1] + CF3B * lin[iin+0] + CF3C * lin[iin-1]); // coeff in reverse order now
				else // normal second pixel (0*00)
					lout[iout+1] = CUCF(CF4A * lin[iin-1] + CF4B * lin[iin+0] + CF4C * lin[iin+1] + CF4D * lin[iin+2]); // offset=iin+1
			}
			else if(x+i == width-2) // second to last pixel (00*0) (3-wide image already handled in if(x+i==1))
			{
				lout[iout+0] = CUCF(CF4D * lin[iin-2] + CF4C * lin[iin-1] + CF4B * lin[iin+0] + CF4A * lin[iin+1]); // offset=iin+1
				lout[iout+1] = CUCF(CF3A * lin[iin+1] + CF3B * lin[iin+0] + CF3C * lin[iin-1]); // coeff in reverse order now
			}
			else if(x+i == width-1) // last pixel (000*)
			{
				lout[iout+0] = CUCF(CF3X * lin[iin-0] + CF3Y * lin[iin-1] + CF3Z * lin[iin-2]);
				lout[iout+1] = CUCF(CF2A * lin[iin-0] + CF2B * lin[iin-1]);
			}
			else // normal middle pixels (...00*00...)
			{
				lout[iout+0] = CUCF(CF4D * lin[iin-2] + CF4C * lin[iin-1] + CF4B * lin[iin+0] + CF4A * lin[iin+1]); // offset=iin+1
				lout[iout+1] = CUCF(CF4A * lin[iin-1] + CF4B * lin[iin+0] + CF4C * lin[iin+1] + CF4D * lin[iin+2]); // offset=iin+1
			}
		}
	}
	// TODO
	//width *= 2;
	//stride = width;
	// lin and lout should be different arrays of memory with the correct dimensions
}

/// Made to be called one thread every 4 vertical input pixels;
///   each thread produces 8 vertical pixels.
/// Note: cache danger!
/// @param[in]  width width of input component, not multiple of 8 / possibly halved
/// @param[in]  height height of input component, not multiple of 8 / possibly halved
/// @param[in]  stride real width of input component pixels, multiple of 8 (if applying for the first time to component, else =width)
/// @param[in]  cin component->pixels
/// @param[out] cout component->pixels double the width
__global__ void njCudaUpsampleV(unsigned char* cin, unsigned char* cout, int width, int height, int stride) {
	const int w = width, s1 = stride, s2 = s1 + s1; // stride, double stride (oss after UpsampleH() stride=width)
	int x = blockIdx.x*blockDim.x + threadIdx.x;       // original pixel x
	int y = (blockIdx.y*blockDim.y + threadIdx.y)*4;   // original pixel y (one thread every 4 pixels in vertical)
	int iin = stride*y+x;
	int iout = stride*y*2 + x; // two output rows for each input row (y), but only one output pixel per input pixel
	int i;
	//out = (unsigned char*) njAllocMem((c->width * c->height) << 1);
	//printf("UpsampleV x=%d y=%d, w=%d, h=%d, str=%d, in %08lx out %08lx\n", x, y, width, height, stride, (unsigned long) cin, (unsigned long) cout);
	if(x < width)
	{
		for(i=0; i<4 && y+i<height; i++, iin+=s1, iout+=2*width) // elaborate (4px in, 8px out) for each thread, stopping at the end of img
		{
			//if(iout+1 >= width*height*2) // TODO rimuovere
			//	printf("UpsampleV iout %d out of bounds x=%d y=%d, w=%d, h=%d, str=%d, in %08lx out %08lx\n", iout, x, y, width, height, stride, (unsigned long) cin, (unsigned long) cout);
			//if(iin >= stride*height)
			//	printf("UpsampleV iin %d out of bounds x=%d y=%d, w=%d, h=%d, str=%d, in %08lx out %08lx\n", iin, x, y, width, height, stride, (unsigned long) cin, (unsigned long) cout);

			if(y+i == 0) // first pixel (*000)
			{
				cout[iout  ] = CUCF(CF2A * cin[iin] + CF2B * cin[iin+s1]);
				cout[iout+w] = CUCF(CF3X * cin[iin] + CF3Y * cin[iin+s1] + CF3Z * cin[iin+s2]);
			}
			else if(y+i == 1) // second pixel (0*00)
			{
				cout[iout  ] = CUCF(CF3A * cin[iin-s1] + CF3B * cin[iin+0] + CF3C * cin[iin+s1]);     // (offset = -1 ?)
				if(x+i == height-2) // second pixel is also second to last (0*0) (image ends right after it started: 3 column wide)
					cout[iout+w] = CUCF(CF3A * cin[iin+s1] + CF3B * cin[iin   ] + CF3C * cin[iin-s1]);
				else // normal second pixel (0*00)
					cout[iout+w] = CUCF(CF4A * cin[iin-s1] + CF4B * cin[iin +0] + CF4C * cin[iin+s1] + CF4D * cin[iin+s2]);
			}
			else if(y+i == height-2) // second to last pixel (00*0) (3-wide image already handled in if(x+i==1))
			{
				cout[iout  ] = CUCF(CF4D * cin[iin-s2] + CF4C * cin[iin-s1] + CF4B * cin[iin +0] + CF4A * cin[iin+s1]);
				cout[iout+w] = CUCF(CF3A * cin[iin+s1] + CF3B * cin[iin +0] + CF3C * cin[iin-s1]);
			}
			else if(y+i == height-1) // last pixel (000*)
			{
				cout[iout  ] = CUCF(CF3X * cin[iin-0] + CF3Y * cin[iin-s1] + CF3Z * cin[iin-s2]);
				cout[iout+w] = CUCF(CF2A * cin[iin-0] + CF2B * cin[iin-s1]);
			}
			else // normal middle pixels (...00*00...)
			{
				cout[iout  ] = CUCF(CF4D * cin[iin-s2] + CF4C * cin[iin-s1] + CF4B * cin[iin +0] + CF4A * cin[iin+s1]);
				cout[iout+w] = CUCF(CF4A * cin[iin-s1] + CF4B * cin[iin +0] + CF4C * cin[iin+s1] + CF4D * cin[iin+s2]);
			}
		}
	}
	//c->height <<= 1;
	//c->stride = c->width;
	//c->pixels = out;
}

NJ_INLINE void njUpsampleH(nj_component_t* c) {
	const int xmax = c->width - 3;
	unsigned char *out, *lin, *lout;
	int x, y;
	out = (unsigned char*) njAllocMem((c->width * c->height) << 1);
	if (!out) njThrow(NJ_OUT_OF_MEM);
	lin = c->pixels;
	lout = out;
	for (y = c->height;  y;  --y) {
		lout[0] = CF(CF2A * lin[0] + CF2B * lin[1]);
		lout[1] = CF(CF3X * lin[0] + CF3Y * lin[1] + CF3Z * lin[2]);
		lout[2] = CF(CF3A * lin[0] + CF3B * lin[1] + CF3C * lin[2]);
		for (x = 0;  x < xmax;  ++x) {
			lout[(x << 1) + 3] = CF(CF4A * lin[x] + CF4B * lin[x + 1] + CF4C * lin[x + 2] + CF4D * lin[x + 3]);
			lout[(x << 1) + 4] = CF(CF4D * lin[x] + CF4C * lin[x + 1] + CF4B * lin[x + 2] + CF4A * lin[x + 3]);
		}
		lin += c->stride;
		lout += c->width << 1;
		lout[-3] = CF(CF3A * lin[-1] + CF3B * lin[-2] + CF3C * lin[-3]);
		lout[-2] = CF(CF3X * lin[-1] + CF3Y * lin[-2] + CF3Z * lin[-3]);
		lout[-1] = CF(CF2A * lin[-1] + CF2B * lin[-2]);
	}
	c->width <<= 1;
	c->stride = c->width;
	njFreeMem((void*)c->pixels);
	c->pixels = out;
}

NJ_INLINE void njUpsampleV(nj_component_t* c) {
	const int w = c->width, s1 = c->stride, s2 = s1 + s1;
	unsigned char *out, *cin, *cout;
	int x, y;
	out = (unsigned char*) njAllocMem((c->width * c->height) << 1);
	if (!out) njThrow(NJ_OUT_OF_MEM);
	for (x = 0;  x < w;  ++x) {
		cin = &c->pixels[x];
		cout = &out[x];
		*cout = CF(CF2A * cin[0] + CF2B * cin[s1]);  cout += w;
		*cout = CF(CF3X * cin[0] + CF3Y * cin[s1] + CF3Z * cin[s2]);  cout += w;
		*cout = CF(CF3A * cin[0] + CF3B * cin[s1] + CF3C * cin[s2]);  cout += w;
		cin += s1;
		for (y = c->height - 3;  y;  --y) {
			*cout = CF(CF4A * cin[-s1] + CF4B * cin[0] + CF4C * cin[s1] + CF4D * cin[s2]);  cout += w;
			*cout = CF(CF4D * cin[-s1] + CF4C * cin[0] + CF4B * cin[s1] + CF4A * cin[s2]);  cout += w;
			cin += s1;
		}
		cin += s1;
		*cout = CF(CF3A * cin[0] + CF3B * cin[-s1] + CF3C * cin[-s2]);  cout += w;
		*cout = CF(CF3X * cin[0] + CF3Y * cin[-s1] + CF3Z * cin[-s2]);  cout += w;
		*cout = CF(CF2A * cin[0] + CF2B * cin[-s1]);
	}
	c->height <<= 1;
	c->stride = c->width;
	njFreeMem((void*) c->pixels);
	c->pixels = out;
}

#else

NJ_INLINE void njUpsample(nj_component_t* c) {
	int x, y, xshift = 0, yshift = 0;
	unsigned char *out, *lin, *lout;
	while (c->width < nj.width) { c->width <<= 1; ++xshift; }
	while (c->height < nj.height) { c->height <<= 1; ++yshift; }
	out = (unsigned char*) njAllocMem(c->width * c->height);
	if (!out) njThrow(NJ_OUT_OF_MEM);
	lin = c->pixels;
	lout = out;
	for (y = 0;  y < c->height;  ++y) {
		lin = &c->pixels[(y >> yshift) * c->stride];
		for (x = 0;  x < c->width;  ++x)
			lout[x] = lin[x >> xshift];
		lout += c->width;
	}
	c->stride = c->width;
	njFreeMem((void*) c->pixels);
	c->pixels = out;
}

#endif

#define PX_PER_THREAD 16

/// Expects to be called one thread for every PX_PER_THREAD horizontal pixels
__global__ void nj_ycbcr_to_rgb(
	unsigned char* py, unsigned char* pcb, unsigned char* pcr,
	int ystride, int cbstride, int crstride,
	unsigned char* rgbout, int width, int height
)
{
	int i;
	//unsigned char *py, *pcb, *pcr;
	int vy, vcb, vcr;
	int x = (blockIdx.x*blockDim.x + threadIdx.x)*PX_PER_THREAD;  // original pixel x (one thread every PX_PER_THREAD pixels in horizontal)
	int y = blockIdx.y*blockDim.y + threadIdx.y;                  // original pixel y
	
	// if(x==0 && y==0)
	// {
	// 	for(i=0; i<PX_PER_THREAD; i++)
	// 		printf("before nj_ycbcr_to_rgb: %3d %3d %3d\n", iny[i], incb[i], incr[i]);
	// }

	if(y < height)
	{
		//if(x==0 && y==0)
		//	printf("nj_ycbcr_to_rgb x=%d y=%d w=%d h=%d\n", x, y, width, height);

		// find starting pointers
		// Aritmetica sui puntatori non era il problema
		//py   = iny  + ystride  * y + x; // single component: one byte each
		//pcb  = incb + cbstride * y + x;
		//pcr  = incr + crstride * y + x;
		
		py   += ystride  * y + x; // single component: one byte each
		pcb  += cbstride * y + x;
		pcr  += crstride * y + x;
		rgbout += (width * y + x) *3; // rgb: 3 byte each


		// convert (up to) PX_PER_THREAD pixels in this thread
		for(i=0; i<PX_PER_THREAD && x < width; i++, x++, py++, pcb++, pcr++, rgbout+=3)
		{
			vy  = *py  << 8;
			vcb = *pcb - 128;
			vcr = *pcr - 128;
			//vy  = iny [ystride  * y + x] << 8;
			//vcb = incb[cbstride * y + x] - 128;
			//vcr = incr[crstride * y + x] - 128;
			// if(x<16 && y==0)
			// 	printf("nj_ycbcr_to_rgb x=%3d y=%3d YCbCr: (%3d %3d %3d) %5d %5d %5d, PTR: %08lx, %08lx, %08lx, rgbout: %08lx\n", x, y, *py, *pcb, *pcr, vy, vcb, vcr, (unsigned long) py, (unsigned long) pcb, (unsigned long) pcr, (unsigned long) rgbout);
			// 	//printf("nj_ycbcr_to_rgb x=%3d y=%3d YCbCr: %5d %5d %5d, PTR: %08lx, %08lx, %08lx, rgbout: %08lx\n", x, y, vy, vcb, vcr, (unsigned long) rgbout);
			rgbout[0] = njCudaClip((vy             + 359 * vcr + 128) >> 8);
			rgbout[1] = njCudaClip((vy -  88 * vcb - 183 * vcr + 128) >> 8);
			rgbout[2] = njCudaClip((vy + 454 * vcb             + 128) >> 8);
			//rgbout[0 + (width * y + x) *3] = njCudaClip((vy             + 359 * vcr + 128) >> 8); // TODO provare senza parentesi come dice Fulvio (width tipo stride)
			//rgbout[1 + (width * y + x) *3] = njCudaClip((vy -  88 * vcb - 183 * vcr + 128) >> 8);
			//rgbout[2 + (width * y + x) *3] = njCudaClip((vy + 454 * vcb             + 128) >> 8);
			//rgbout[0 + (width * y + x) *3] = vy; // TODO rimuovere
			//rgbout[2 + (width * y + x) *3] = vy;
			//rgbout[1 + (width * y + x) *3] = vy;
		}
	}
}

NJ_INLINE void njCudaConvert(void) {
	int i;
	nj_component_t* c;
	unsigned char* newvec;
	//dim3 dimBlock (16, 16);	//roundup 
	//dim3 dimGrid ((n_blocks + 255)/256, 1);
	//dim3 dimGridCbCr ((CbCr_blocks + 255)/256, 1);

	//for(i=0; i<16;i++)
	//	printf("Prima, YCbCr: %3d %3d %3d\n", nj.comp[0].pixels[i], nj.comp[1].pixels[i], nj.comp[2].pixels[i]);
	
	if(failed(hipMalloc((void**)&(nj.curgb), nj.width * nj.height * 3))) // temporary memcpy to try this CUDA version
		printf("malloc curgb failed\n");
	
	for (i = 0, c = nj.comp;  i < nj.ncomp;  ++i, ++c) {
		//printf("component %d: stride %d, mbheight %d, ssy %d\n", i, c->stride, nj.mbheight, c->ssy);

		// if(failed(hipMalloc((void**)&(c->cupixels), c->stride * nj.mbheight * c->ssy << 3))) // temporary memcpy to try this CUDA version, moved to njCudaDecodeScan()
		// 	printf("malloc component failed\n");
		// if(failed(hipMemcpy( c->cupixels, c->pixels, c->stride * nj.mbheight * c->ssy << 3, hipMemcpyHostToDevice )))
		// 	printf("initial memcpy component failed\n");
		
		// if(failed(hipDeviceSynchronize())) // ==================================
		// 	printf("sync after UpsampleH component %d failed.\n", i);
		//printf("component %d: pix %08lx cupix %08lx\n", i, (unsigned long) c->pixels, (unsigned long) c->cupixels);

		//#if NJ_CHROMA_FILTER
			while ((c->width < nj.width) || (c->height < nj.height)) {
				if (c->width < nj.width)
				{
					if(failed(hipMalloc((void**)&newvec, c->width * c->height * 2)))
						printf("malloc newvec component horizontal realloc failed\n");

					dim3 dimBlock (8, 32);	// thread per grid cell: 8x32=256 thread per grid
					dim3 dimGrid (((nj.width+3)/4 + 7)/8, (nj.height+31)/32); // grid size

					//printf("UpsampleH dimGrid %dx%d dimBlock %dx%d\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);
					//printf("component %d: pix %08lx cupix %08lx, newvec %08lx 2a print\n", i, (unsigned long) c->pixels, (unsigned long) c->cupixels, (unsigned long) newvec);
					
					njCudaUpsampleH<<<dimGrid, dimBlock>>>(c->cupixels, newvec, c->width, c->height, c->stride); // TODO call it better
					
					if (failed(hipPeekAtLastError()))
						printf("error UpsampleH component %d failed\n", i);
					if(failed(hipDeviceSynchronize())) // ==================================
						printf("sync after UpsampleH component %d failed.\n", i);
					if(failed(hipFree(c->cupixels)))
						printf("free cupixels UpsampleH component %d failed\n", i);
					c->cupixels = newvec;
					c->width *= 2;
					c->stride = c->width;
					//c->pixels = (unsigned char *) realloc(c->pixels, c->stride*c->height); // TODO rimuovere
				}
				njCheckError();
				if (c->height < nj.height)
				{
					if(failed(hipMalloc((void**)&newvec, c->width * c->height * 2)))
						printf("malloc newvec component vertical realloc failed\n");

					dim3 dimBlock (32, 8);	// thread per grid cell
					dim3 dimGrid ((nj.width + 31)/32, ((nj.height+3)/4 + 7)/8); // grid size

					//printf("UpsampleV dimGrid %dx%d dimBlock %dx%d\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);
					//printf("component %d: pix %08lx cupix %08lx, newvec %08lx 3a print\n", i, (unsigned long) c->pixels, (unsigned long) c->cupixels, (unsigned long) newvec);

					njCudaUpsampleV<<<dimGrid, dimBlock>>>(c->cupixels, newvec, c->width, c->height, c->stride); // TODO call it better

					if(failed(hipPeekAtLastError()))
						printf("error UpsampleV component %d failed\n", i);
					if(failed(hipDeviceSynchronize())) // ==================================
						printf("sync after UpsampleV component %d failed.\n", i);
					
					if(failed(hipFree(c->cupixels)))
						printf("free cupixels UpsampleV component %d failed\n", i);
					c->cupixels = newvec;
					c->height *= 2;
					c->stride = c->width;
					//c->pixels = (unsigned char *) realloc(c->pixels, c->stride*c->height); // TODO rimuovere
				}
				njCheckError();
			}
		//#else
		//	if ((c->width < nj.width) || (c->height < nj.height))
		//		njUpsample(c);
		//#endif
		if ((c->width < nj.width) || (c->height < nj.height)) njThrow(NJ_INTERNAL_ERR);

		if (failed(hipPeekAtLastError()))
        	printf("peek last error failed alla fine del ciclo component %d\n", i);
		//memset(c->pixels, 0, c->stride * c->height); // TODO rimuovere solo diagnostica
		//printf("copy of %d byte component %d.\n", c->stride * c->height, i);
		//if(failed(hipMemcpy( c->pixels, c->cupixels, c->stride * c->height, hipMemcpyDeviceToHost ))) // TODO rimuovere
		//	printf("final temporary memcpy component failed, pixels=%08lx, cupix=%08lx\n", (unsigned long) c->pixels, (unsigned long) c->cupixels);
	} // end foreach component

	//for(i=0; i<16;i++)
	//	printf("dopo subsample, YCbCr: %3d %3d %3d\n", nj.comp[0].pixels[i], nj.comp[1].pixels[i], nj.comp[2].pixels[i]);
	
	if (nj.ncomp == 3) {
		// convert to RGB (8-stride may be already removed either horizontally or vertically in Upsample)

		dim3 dimBlock (8, 32);	// thread per grid cell: 8x32=256 thread per grid
		dim3 dimGrid (((nj.width+PX_PER_THREAD-1)/PX_PER_THREAD + 7)/8, (nj.height+31)/32);
		//dim3 dimBlock (1, 8);	// thread per grid cell: 8x32=256 thread per grid
		//dim3 dimGrid (1, 1);

		//printf("nj_ycbcr_to_rgb block %dx%d, dimGrid %dx%d\n", dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y);
		//printf("  cupixels: %08lx, %08lx, %08lx\n", (unsigned long) nj.comp[0].cupixels, (unsigned long) nj.comp[1].cupixels, (unsigned long) nj.comp[2].cupixels);
		//printf("  strides:  %8d, %8d, %8d\n", nj.comp[0].stride, nj.comp[1].stride, nj.comp[2].stride);
		//printf("  curgb:    %08lx, w=%d h=%d\n", (unsigned long) nj.curgb, nj.width, nj.height);

		nj_ycbcr_to_rgb <<<dimGrid, dimBlock>>>( // TODO chiamare meglio: stream
			nj.comp[0].cupixels, nj.comp[1].cupixels, nj.comp[2].cupixels,
			nj.comp[0].stride, nj.comp[1].stride, nj.comp[2].stride,
			nj.curgb, nj.width, nj.height
		);

		if(failed(hipPeekAtLastError()))
			printf("error nj_ycbcr_to_rgb failed\n");
		if(failed(hipDeviceSynchronize())) // ==================================
			printf("sync after nj_ycbcr_to_rgb failed.\n");
		
		if(failed(hipMemcpy(nj.rgb, nj.curgb, nj.width * nj.height * 3, hipMemcpyDeviceToHost)))
			printf("memcpy rgb d2host failed\n");
		hipFree(nj.curgb);
	} else if (nj.comp[0].width != nj.comp[0].stride) {
		// grayscale -> only remove 8-stride
		unsigned char *pin = &nj.comp[0].pixels[nj.comp[0].stride];
		unsigned char *pout = &nj.comp[0].pixels[nj.comp[0].width];
		int y;
		for (y = nj.comp[0].height - 1;  y;  --y) {
			njCopyMem(pout, pin, nj.comp[0].width);
			pin += nj.comp[0].stride;
			pout += nj.comp[0].width;
		}
		nj.comp[0].stride = nj.comp[0].width;
	}
}

NJ_INLINE void njConvert(void) {
	int i;
	nj_component_t* c;
	for (i = 0, c = nj.comp;  i < nj.ncomp;  ++i, ++c) {
		#if NJ_CHROMA_FILTER
			while ((c->width < nj.width) || (c->height < nj.height)) {
				if (c->width < nj.width) njUpsampleH(c);
				njCheckError();
				if (c->height < nj.height) njUpsampleV(c);
				njCheckError();
			}
		#else
			if ((c->width < nj.width) || (c->height < nj.height))
				njUpsample(c);
		#endif
		if ((c->width < nj.width) || (c->height < nj.height)) njThrow(NJ_INTERNAL_ERR);
	}
	if (nj.ncomp == 3) {
		// convert to RGB (8-stride may be already removed either horizontally or vertically in Upsample)
		int x, yy;
		unsigned char *prgb = nj.rgb;
		const unsigned char *py  = nj.comp[0].pixels;
		const unsigned char *pcb = nj.comp[1].pixels;
		const unsigned char *pcr = nj.comp[2].pixels;
		for (yy = nj.height;  yy;  --yy) {
			for (x = 0;  x < nj.width;  ++x) {
				int y = py[x] << 8;
				int cb = pcb[x] - 128;
				int cr = pcr[x] - 128;
				*prgb++ = njClip((y            + 359 * cr + 128) >> 8);
				*prgb++ = njClip((y -  88 * cb - 183 * cr + 128) >> 8);
				*prgb++ = njClip((y + 454 * cb            + 128) >> 8);
			}
			py += nj.comp[0].stride;
			pcb += nj.comp[1].stride;
			pcr += nj.comp[2].stride;
		}
	} else if (nj.comp[0].width != nj.comp[0].stride) {
		// grayscale -> only remove 8-stride
		unsigned char *pin = &nj.comp[0].pixels[nj.comp[0].stride];
		unsigned char *pout = &nj.comp[0].pixels[nj.comp[0].width];
		int y;
		for (y = nj.comp[0].height - 1;  y;  --y) {
			njCopyMem(pout, pin, nj.comp[0].width);
			pin += nj.comp[0].stride;
			pout += nj.comp[0].width;
		}
		nj.comp[0].stride = nj.comp[0].width;
	}
}

void njInit(int use_cuda) {
	int i;

	njFillMem(&nj, 0, sizeof(nj_context_t));
	nj.use_cuda = use_cuda;

	if(nj.use_cuda)
	{
		for(i=0; i<NSTR; i++)
		{
			printf("doing hipStreamCreate(%016lx) stream %d ... ", (unsigned long) &(nj.custreams[i]), i); // TODO togliere debug
			if(failed(hipStreamCreate(&(nj.custreams[i]))))
				printf("failed hipStreamCreate stream %d\n", i);
			printf("done hipStreamCreate(%016lx) stream %d .\n", (unsigned long) nj.custreams[i], i);
		}
	}
}

void njDone(void) {
	int i;
	if(nj.use_cuda)
	{
		for(i=0; i<NSTR; i++)
		{
			printf("doing hipStreamDestroy(%016lx) stream %d ... ", (unsigned long) &(nj.custreams[i]), i); // TODO togliere debug
			if(failed(hipStreamDestroy(nj.custreams[i])))
				printf("failed hipStreamDestroy stream %d\n", i);
			printf("done hipStreamCreate(%016lx) stream %d .\n", (unsigned long) nj.custreams[i], i);
		}
	}
	for (i = 0;  i < 3;  ++i) // TODO non dovrebbe essere i < nj.ncomp?
	{
		if (nj.comp[i].pixels) njFreeMem((void*) nj.comp[i].pixels);
		if (nj.comp[i].intpixels) njFreeMem((void*) nj.comp[i].intpixels);
	}
	if (nj.rgb) njFreeMem((void*) nj.rgb);
	njInit(nj.use_cuda);
}

/// Main call to decompress a JPEG
nj_result_t njDecode(const void* jpeg, const int size) {
	njDone();
	if(nj.use_cuda)
		hipDeviceReset();
	nj.pos = (const unsigned char*) jpeg;
	nj.size = size & 0x7FFFFFFF;
	printf("use_cuda=%d size=%d magic=%02x %02x\n", nj.use_cuda, nj.size, (unsigned) nj.pos[0], (unsigned) nj.pos[1]);
	if (nj.size < 2) return NJ_NO_JPEG;
	if ((nj.pos[0] ^ 0xFF) | (nj.pos[1] ^ 0xD8)) return NJ_NO_JPEG;
	njSkip(2);
	while (!nj.error) {
		if ((nj.size < 2) || (nj.pos[0] != 0xFF)) return NJ_SYNTAX_ERROR;
		njSkip(2);
		switch (nj.pos[-1]) {
			case 0xC0: njDecodeSOF();  break;
			case 0xC4: njDecodeDHT();  break;
			case 0xDB: njDecodeDQT();  break;
			case 0xDD: njDecodeDRI();  break;
			case 0xDA:
				if(nj.use_cuda) njCudaDecodeScan();
				else njDecodeScan();
				break; // CUDA mod
			case 0xFE: njSkipMarker(); break;
			default:
				if ((nj.pos[-1] & 0xF0) == 0xE0)
					njSkipMarker();
				else
					return NJ_UNSUPPORTED;
		}
	}
	if (nj.error != __NJ_FINISHED) return nj.error;
	nj.error = NJ_OK;

	if(nj.use_cuda)
		njCudaConvert();
	else
		njConvert();
	return nj.error;
}

int njGetWidth(void)            { return nj.width; }
int njGetHeight(void)           { return nj.height; }
int njIsColor(void)             { return (nj.ncomp != 1); }
unsigned char* njGetImage(void) { return (nj.ncomp == 1) ? nj.comp[0].pixels : nj.rgb; }
int njGetImageSize(void)        { return nj.width * nj.height * nj.ncomp; }

// Call tree:
//
// njDecode()
//   -> njDecodeSOF/DHT/DQT/DRI()
//   -> njDecodeScan()
//     -> njDecodeBlock()
//       -> njGetVLC()
//       -> njRowIDCT()
//       -> njColIDCT()
//   -> njSkipMarker() chissenefrega
//   -> njConvert()
//     -> njUpsample/H/V()
//     -> njClip()
