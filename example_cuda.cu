#include "hip/hip_runtime.h"
// //////////////////////////////////////////////////////////
// how to use TooJpeg: creating a JPEG file
// see https://create.stephan-brumme.com/toojpeg/
// compile: g++ example.cpp toojpeg.cpp -o example -std=c++11

#include "toojpeg.h"

// //////////////////////////////////////////////////////////
// use a C++ file stream
#include <fstream>
#include <time.h>

// output file
std::ofstream myFile("example_cuda.jpg", std::ios_base::out | std::ios_base::binary);

// write a single byte compressed by tooJpeg
void myOutput(unsigned char byte)
{
  myFile << byte;
}

// //////////////////////////////////////////////////////////
int main()
{

	clock_t start, end;
	
	start=clock();

  // 800x600 image
  const auto width  = 8000;
  const auto height = 6000;
  // RGB: one byte each for red, green, blue
  const auto bytesPerPixel = 3;

  // allocate memory
  auto image = new unsigned char[width * height * bytesPerPixel];

  // create a nice color transition (replace with your code)
  for (auto y = 0; y < height; y++)
    for (auto x = 0; x < width; x++)
    {
      // memory location of current pixel
      auto offset = (y * width + x) * bytesPerPixel;

      // red and green fade from 0 to 255, blue is always 127
      image[offset    ] = 255 * x / width;
      image[offset + 1] = 255 * y / height;
      image[offset + 2] = 127;
    }

  // start JPEG compression
  // note: myOutput is the function defined in line 18, it saves the output in example.jpg
  // optional parameters:
  const bool isRGB      = true;  // true = RGB image, else false = grayscale
  const auto quality    = 90;    // compression quality: 0 = worst, 100 = best, 80 to 90 are most often used
  const bool downsample = true; // false = save as YCbCr444 JPEG (better quality), true = YCbCr420 (smaller file)
  const char* comment = "TooJpeg example image"; // arbitrary JPEG comment
  auto ok = TooJpeg::writeJpeg(myOutput, image, width, height, isRGB, quality, downsample, comment);

  delete[] image;

	end=clock();
	
	printf("time: %f", double(end-start) / CLOCKS_PER_SEC);

  // error => exit code 1
  return ok ? 0 : 1;
}
