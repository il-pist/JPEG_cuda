#include "hip/hip_runtime.h"
// //////////////////////////////////////////////////////////
// how to use TooJpeg: creating a JPEG file
// see https://create.stephan-brumme.com/toojpeg/
// compile: g++ example.cpp toojpeg.cpp -o example -std=c++11

#include "toojpeg.h"

// //////////////////////////////////////////////////////////
// use a C++ file stream
#include <fstream>
#include <time.h>

// output file
const char* filename = "example_gray_cuda.jpg";
std::ofstream myFile(filename, std::ios_base::out | std::ios_base::binary);

// write a single byte compressed by TooJpeg
void myOutput(unsigned char byte)
{
  myFile << byte;
}

// //////////////////////////////////////////////////////////
int main()
{
	clock_t start, end;


  // 8000x6000 image
  const auto width  = 8000;
  const auto height = 6000;
  // Grayscale: one byte per pixel
  const auto bytesPerPixel = 1;

  // allocate memory
  auto image = new unsigned char[width * height * bytesPerPixel];

  // create a nice color transition (replace with your code)
  for (auto y = 0; y < height; y++)
    for (auto x = 0; x < width; x++)
    {
      // memory location of current pixel
      auto offset = (y * width + x) * bytesPerPixel;

      // red and green fade from 0 to 255, blue is always 127
      auto red   = 255 * x / width;
      auto green = 255 * y / height;
      image[offset] = (red + green) / 2;;
    }

  // start JPEG compression
  // note: myOutput is the function defined in line 18, it saves the output in example.jpg
  // optional parameters:
  const bool isRGB      = false; // true = RGB image, else false = grayscale
  const auto quality    = 90;    // compression quality: 0 = worst, 100 = best, 80 to 90 are most often used
  const bool downsample = false; // false = save as YCbCr444 JPEG (better quality), true = YCbCr420 (smaller file)
  const char* comment   = "TooJpeg example image"; // arbitrary JPEG comment

  start=clock();

  auto ok = TooJpeg::writeJpeg(myOutput, image, width, height, isRGB, quality, downsample, comment);

  delete[] image;

  end=clock();

  printf("time: %f\n\n", double(end-start) / CLOCKS_PER_SEC);


  // error => exit code 1
  return ok ? 0 : 1;
}
